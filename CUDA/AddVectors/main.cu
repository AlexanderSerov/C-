#include "hip/hip_runtime.h"
/*
 * Set up environment for compiling cuda:
 * 1) Install compatible toolchain VS 2017 or 2019. Install it through separate installer to get full functionality.
 * 2) You can compile in Command Prompt now using nvcc. For using Powershell x64 follow instructions in Workflow
 * 		document on Drive.
 * 3) To be able VS toolset locate you cuda installation you need copy nvidia extension for VS to VS toolset.
 * 		Details: https://github.com/mitsuba-renderer/mitsuba2/issues/103#issuecomment-618378963
 * 4) If you use CLion you just need to specify desired, preconfigured toolset.
 * */
#include <iostream>
#include <memory>
#include <vector>
#include <hip/hip_runtime.h>


// Kernel function to add the elements of two arrays
__global__
void add(int n, const float *x, float *y)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i += stride)
		y[i] = x[i] + y[i];
}

template <typename T>
std::unique_ptr<T[], hipError_t(*)(void *)> vector_to_cuda_memory(const std::vector<T>& vec) {
    // Calculate the size of the CUDA memory in bytes
    size_t cuda_size = vec.size() * sizeof(T);

    // Allocate CUDA memory using hipMalloc
    T* cuda_ptr = nullptr;
    hipError_t status = hipMalloc(&cuda_ptr, cuda_size);
    if (status != hipSuccess) {
        throw std::runtime_error("hipMalloc failed");
    }

    // Copy the vector's data to CUDA memory using hipMemcpy
    status = hipMemcpy(cuda_ptr, vec.data(), cuda_size, hipMemcpyHostToDevice);
    if (status != hipSuccess) {
        hipFree(cuda_ptr);
        throw std::runtime_error("hipMemcpy failed");
    }

    // Create a unique_ptr to manage the CUDA memory with a custom deleter
	return std::unique_ptr<T[], hipError_t(*)(void *)>(cuda_ptr, hipFree);
}


int main()
{
	std::vector<int> vec {1,2,3};
	const int a[] {1,2,3};
	const int b[] {4,5,6};
	const int c[] {7,8,9};
	const std::vector<const void*> void_vec{a, b, c};
	{
		auto ptr = vector_to_cuda_memory(vec);
		auto ptr2 = vector_to_cuda_memory(void_vec);
	}


///////////////////////////////////////////////////////////////////

	int N = 1<<20;
	float *x, *y;

	// Allocate Unified Memory – accessible from CPU or GPU
	hipMallocManaged(&x, N*sizeof(float));
	hipMallocManaged(&y, N*sizeof(float));

	// initialize x and y arrays on the host
	for (int i = 0; i < N; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

	// Prefetch the data to the GPU
	int device = -1;
	hipGetDevice(&device);
	std::cout << "device: " << device << std::endl;
	hipMemPrefetchAsync(x, N*sizeof(float), device, nullptr);
	hipMemPrefetchAsync(y, N*sizeof(float), device, nullptr);

	// Run kernel on 1M elements on the GPU
	int blockSize = 256;
	int gridSize = (N / blockSize) + 1;
	add<<<gridSize, blockSize>>>(N, x, y);

	// Prefetch the data to the CPU
	hipMemPrefetchAsync(y, N*sizeof(float), hipCpuDeviceId, nullptr);

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();

	// Check for errors (all values should be 3.0f)
	float maxError = 0.0f;
	for (int i = 0; i < N; i++)
		maxError = fmax(maxError, fabs(y[i]-3.0f));
	std::cout << "Max error: " << maxError << std::endl;

	// Free memory
	hipFree(x);
	hipFree(y);

	// For cuda-memcheck work correctly
	hipDeviceReset();

	return 0;
}